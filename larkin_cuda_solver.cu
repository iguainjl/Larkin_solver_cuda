
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hipfft/hipfft.h>
#include <getopt.h>
#include <vector>
#include <string>
#include <iostream>
#include <fstream>

#define BLOCK_SIZE 256

// CUDA error check
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess) {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

// Kernel: initialize RNG states
__global__ void init_rng(hiprandState *state, unsigned long seed, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) hiprand_init(seed, idx, 0, &state[idx]);
}

// Kernel: add noise η(x) dt
__global__ void add_noise(double *h, hiprandState *state, double Delta, double dt, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        double r = hiprand_normal_double(&state[idx]);
        h[idx] += sqrt(dt) * Delta * r;
    }
}

// Kernel: compute derivative u = ∂x h
__global__ void compute_u(const double *h, double *u, double dx, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        int ip = (idx+1) % N;
        int im = (idx-1+N) % N;
        u[idx] = (h[ip] - h[im]) / (2.0*dx);
    }
}

// Kernel: compute nonlinear flux F(u) = sign(u) |u|^(2n-1)
__global__ void compute_flux(const double *u, double *F, int n, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        double val = u[idx];
        double s = (val > 0) - (val < 0);
        F[idx] = s * pow(fabs(val), 2*n - 1);
    }
}

// Kernel: compute derivative of flux dF/dx
__global__ void compute_dFdx(const double *F, double *dFdx, double dx, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        int ip = (idx+1) % N;
        int im = (idx-1+N) % N;
        dFdx[idx] = (F[ip] - F[im]) / (2.0*dx);
    }
}

// Kernel: Euler update
__global__ void euler_update(double *h, const double *dFdx, double dt, double nu, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        h[idx] += dt * nu * dFdx[idx];
    }
}

// Kernel: count zero crossings of u
__global__ void count_zeros(const double *u, int *counts, int N) {
    __shared__ int local[BLOCK_SIZE];
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;
    int val = 0;
    if (idx < N) {
        int ip = (idx+1) % N;
        if (u[idx] == 0.0 || u[ip] == 0.0) val = 1;
        else if ((u[idx] > 0 && u[ip] < 0) || (u[idx] < 0 && u[ip] > 0)) val = 1;
    }
    local[tid] = val;
    __syncthreads();
    for (int s = blockDim.x/2; s > 0; s >>= 1) {
        if (tid < s) local[tid] += local[tid + s];
        __syncthreads();
    }
    if (tid == 0) counts[blockIdx.x] = local[0];
}

// Host: sum zero crossings
int get_zero_crossings(const double *u_d, int N) {
    int blocks = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
    int *d_counts, *h_counts = new int[blocks];
    gpuErrchk(hipMalloc(&d_counts, blocks * sizeof(int)));
    count_zeros<<<blocks,BLOCK_SIZE>>>(u_d, d_counts, N);
    gpuErrchk(hipMemcpy(h_counts, d_counts, blocks*sizeof(int), hipMemcpyDeviceToHost));
    int total = 0;
    for (int i=0; i<blocks; i++) total += h_counts[i];
    hipFree(d_counts);
    delete [] h_counts;
    return total;
}

// FFT-based structure factor
void compute_structure_factor(double *u_d, int N, double L, const std::string &fname) {
    hipfftHandle plan;
    hipfftDoubleReal *data_d;
    hipfftDoubleComplex *fft_d;
    size_t workSize;
    int Nc = N/2 + 1;

    gpuErrchk(hipMalloc(&data_d, N*sizeof(double)));
    gpuErrchk(hipMemcpy(data_d, u_d, N*sizeof(double), hipMemcpyDeviceToDevice));
    gpuErrchk(hipMalloc(&fft_d, Nc*sizeof(hipfftDoubleComplex)));

    hipfftPlan1d(&plan, N, HIPFFT_D2Z, 1);
    hipfftExecD2Z(plan, data_d, fft_d);

    std::vector<double> Su(Nc);
    hipfftDoubleComplex *fft_h = new hipfftDoubleComplex[Nc];
    gpuErrchk(hipMemcpy(fft_h, fft_d, Nc*sizeof(hipfftDoubleComplex), hipMemcpyDeviceToHost));
    for (int k=0; k<Nc; k++) {
        double re = fft_h[k].x;
        double im = fft_h[k].y;
        Su[k] = (re*re + im*im) / N;
    }

    std::ofstream fout(fname);
    double dk = 2.0*M_PI/L;
    for (int k=0; k<Nc; k++) {
        fout << k*dk << " " << Su[k] << "\n";
    }
    fout.close();

    hipfftDestroy(plan);
    hipFree(data_d);
    hipFree(fft_d);
    delete [] fft_h;
}

// -------------------- MAIN --------------------

int main(int argc, char** argv) {
    // Default parameters
    int N = 1024;
    double L = 100.0;
    int n = 2;
    double tmax = 10.0;
    double dt = 0.01;
    double nu = 1.0;
    double Delta = 0.1;
    unsigned long seed = 1234;
    std::string rhoFile = "rho_vs_t.dat";
    std::string SuFile  = "Su_final.dat";

    static struct option long_options[] = {
        {"N", required_argument, 0, 'N'},
        {"L", required_argument, 0, 'L'},
        {"n", required_argument, 0, 'n'},
        {"tmax", required_argument, 0, 't'},
        {"dt", required_argument, 0, 'd'},
        {"nu", required_argument, 0, 'u'},
        {"Delta", required_argument, 0, 'D'},
        {"seed", required_argument, 0, 's'},
        {"out", required_argument, 0, 'o'},
        {"outSu", required_argument, 0, 'S'},
        {0,0,0,0}
    };

    int opt, idx;
    while ((opt = getopt_long(argc, argv, "", long_options, &idx)) != -1) {
        switch(opt) {
            case 'N': N = atoi(optarg); break;
            case 'L': L = atof(optarg); break;
            case 'n': n = atoi(optarg); break;
            case 't': tmax = atof(optarg); break;
            case 'd': dt = atof(optarg); break;
            case 'u': nu = atof(optarg); break;
            case 'D': Delta = atof(optarg); break;
            case 's': seed = atol(optarg); break;
            case 'o': rhoFile = optarg; break;
            case 'S': SuFile = optarg; break;
        }
    }

    double dx = L / N;
    int steps = int(tmax/dt);

    // Allocate device memory
    double *h_d, *u_d, *F_d, *dFdx_d;
    gpuErrchk(hipMalloc(&h_d, N*sizeof(double)));
    gpuErrchk(hipMalloc(&u_d, N*sizeof(double)));
    gpuErrchk(hipMalloc(&F_d, N*sizeof(double)));
    gpuErrchk(hipMalloc(&dFdx_d, N*sizeof(double)));
    gpuErrchk(hipMemset(h_d, 0, N*sizeof(double)));

    // RNG states
    hiprandState *rng_d;
    gpuErrchk(hipMalloc(&rng_d, N*sizeof(hiprandState)));
    int blocks = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
    init_rng<<<blocks,BLOCK_SIZE>>>(rng_d, seed, N);

    std::ofstream fout(rhoFile);
    for (int step=0; step<=steps; step++) {
        double t = step*dt;

        // Compute u, flux, dFdx
        compute_u<<<blocks,BLOCK_SIZE>>>(h_d, u_d, dx, N);
        compute_flux<<<blocks,BLOCK_SIZE>>>(u_d, F_d, n, N);
        compute_dFdx<<<blocks,BLOCK_SIZE>>>(F_d, dFdx_d, dx, N);

        // Update
        euler_update<<<blocks,BLOCK_SIZE>>>(h_d, dFdx_d, dt, nu, N);
        add_noise<<<blocks,BLOCK_SIZE>>>(h_d, rng_d, Delta, dt, N);

        // Zero crossings density
        if (step % 10 == 0) {
            int zeros = get_zero_crossings(u_d, N);
            double rho = (double)zeros / L;
            fout << t << " " << rho << "\n";
        }
    }
    fout.close();

    // Compute structure factor
    compute_structure_factor(u_d, N, L, SuFile);

    // Free
    hipFree(h_d);
    hipFree(u_d);
    hipFree(F_d);
    hipFree(dFdx_d);
    hipFree(rng_d);

    printf("Simulation done. Output written to %s and %s\n", rhoFile.c_str(), SuFile.c_str());
    return 0;
}


